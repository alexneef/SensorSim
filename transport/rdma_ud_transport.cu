#include "hip/hip_runtime.h"
//
// Created by alex on 8/7/20.
//

#include "rdma_ud_transport.cuh"

#include <cstdio>
#include <algorithm>
#include <arpa/inet.h>
#include <iostream>
#include <sys/types.h>
#include <sys/socket.h>
#include <netdb.h>


int get_addr(const char *dst, struct sockaddr *addr)
{
    struct addrinfo *res;
    int ret;
    ret = getaddrinfo(dst, NULL, NULL, &res);
    if (ret)
    {
        fprintf(stderr,"ERROR: getaddrinfo failed - invalid hostname or IP address\n");
        return -1;
    }
    memcpy(addr, res->ai_addr, res->ai_addrlen);
    freeaddrinfo(res);
    return ret;
}

void PrintCMEvent(struct rdma_cm_event *event)
{
    if(event->event == RDMA_CM_EVENT_ADDR_RESOLVED)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ADDR_RESOLVED)\n");
    else if(event->event == RDMA_CM_EVENT_ADDR_RESOLVED)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ADDR_RESOLVED)\n");
    else if(event->event == RDMA_CM_EVENT_ROUTE_RESOLVED)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ROUTE_RESOLVED)\n");
    else if(event->event == RDMA_CM_EVENT_ROUTE_ERROR)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ROUTE_ERROR)\n");
    else if(event->event == RDMA_CM_EVENT_CONNECT_REQUEST)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_CONNECT_REQUEST)\n");
    else if(event->event == RDMA_CM_EVENT_CONNECT_RESPONSE)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_CONNECT_RESPONSE)\n");
    else if(event->event == RDMA_CM_EVENT_CONNECT_ERROR)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_CONNECT_ERROR)\n");
    else if(event->event == RDMA_CM_EVENT_UNREACHABLE)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_UNREACHABLE)\n");
    else if(event->event == RDMA_CM_EVENT_REJECTED)
    {
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_REJECTED)\n");
        fprintf(stderr,"DEBUG: Status(%u)", event->status);
    }
    else if(event->event == RDMA_CM_EVENT_ESTABLISHED)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ESTABLISHED)\n");
    else if(event->event == RDMA_CM_EVENT_DISCONNECTED)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_DISCONNECTED)\n");
    else if(event->event == RDMA_CM_EVENT_DEVICE_REMOVAL)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_DEVICE_REMOVAL)\n");
    else if(event->event == RDMA_CM_EVENT_MULTICAST_JOIN)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_MULTICAST_JOIN)\n");
    else if(event->event == RDMA_CM_EVENT_MULTICAST_ERROR)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_MULTICAST_ERROR)\n");
    else if(event->event == RDMA_CM_EVENT_ADDR_CHANGE)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ADDR_CHANGE)\n");
    else if(event->event == RDMA_CM_EVENT_TIMEWAIT_EXIT)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_TIMEWAIT_EXIT)\n");
    return;
}

void PrintConnectionInfo(rdma_conn_param cParam)
{
    fprintf(stderr, "DEBUG: QPN(%d)\n", cParam.qp_num);
}


RdmaUdTransport::RdmaUdTransport(string srcAddr, int srcPort, string dstAddr, int dstPort, eTransportRole role) {

    s_srcAddr = srcAddr;
    n_srcPort = srcPort;
    s_dstAddr = dstAddr;
    n_dstPort = dstPort;

    // Creating socket file descriptor
    if(RDMACreateChannel() != 0)
    {
        cerr << "Failed Create the RDMA Channel." << endl;
        exit(EXIT_FAILURE);
    }

    if(role == eTransportRole::SENSOR) { //Sensor
        if (RDMAClientInit() != 0) {
            fprintf(stdout, "Exiting - Failed to initialize the Client Side CM Connection.\n");
            exit(EXIT_FAILURE);
        }

        if(RDMACreateQP() != 0)
        {
            //TODO: We should be able to start the processes in any order. Need things to wait and retry.
            fprintf(stdout, "Exiting - Failed to Create Queue Pair, make sure processor is running\n");
            exit(EXIT_FAILURE);
        }

        if(RDMAClientConnect() != 0)
        {
            fprintf(stdout, "Exiting - Failed to establish connection to client\n");
            exit(EXIT_FAILURE);
        }

        /*
        //Initialize the wqe's used for send
        sendWQEs.resize(MSG_BLOCK_SIZE);
        int i = 0; //wr id
        std::for_each(begin(sendWQEs), end(sendWQEs), [&] (ibv_send_wr &wqe) {
           initSendWqe(&wqe, i++);
        });
        */

        //Register the control plane memory region
        mr_controlBuffer = create_MEMORY_REGION(&controlBuffer, MSG_MAX_SIZE);
        memset(controlBuffer, 0xFF, MSG_MAX_SIZE);

        initSendWqe(&controlSendWqe, 0);
        updateSendWqe(&controlSendWqe, &controlBuffer, MSG_MAX_SIZE, mr_controlBuffer);

        sleep(1); //Wait 1 Second or I get a Completion Error TODO: Fix This CP Error

        post_SEND_WQE(&controlSendWqe);
        cout << "Sending first control message" << endl;

        //Wait For Completion
        ibv_wc wc;
        PollCQ(&wc);

    } else { //Processor
        if (RDMAServerInit() != 0) {
            fprintf(stdout, "Exiting - Failed to initialize the Server Side CM Connection.\n");
            exit(EXIT_FAILURE);
        }

        if(RDMAServerConnect() != 0)
        {
            fprintf(stderr, "Exiting - Failed to establish connection with the client\n");
            exit(EXIT_FAILURE);
        }

        /*
        //Register the rcvBuffer memory region
        mr_rcvBuffer = create_MEMORY_REGION(&rcvBuffer, MSG_MAX_SIZE * MSG_BLOCK_SIZE);
        memset(rcvBuffer, 0x00, MSG_MAX_SIZE * MSG_BLOCK_SIZE);

        //Initialize the wqe's used for receiving and point to the MR
        rcvWQEs.resize(MSG_BLOCK_SIZE);
        int i = 0;
        std::for_each(begin(rcvWQEs), end(rcvWQEs), [&] (ibv_recv_wr &wqe) {
            initRecvWqe(&wqe, i);
            //updateRecvWqe(&wqe, &rcvBuffer[i*MSG_MAX_SIZE], MSG_MAX_SIZE, mr_rcvBuffer);
            updateRecvWqe(&wqe, &rcvBuffer[0], MSG_MAX_SIZE * MSG_BLOCK_SIZE, mr_rcvBuffer);
            if(i < rcvWQEs.size()-1) { wqe.next = &rcvWQEs[i+1];} //connect the WQES so we can post at once.
            i++;
        });
         */

        //Register the control plane memory region
        mr_controlBuffer = create_MEMORY_REGION(&controlBuffer, MSG_MAX_SIZE);
        memset(controlBuffer, 0x00, MSG_MAX_SIZE);

        initRecvWqe(&controlRcvWqe, 0);
        updateRecvWqe(&controlRcvWqe, &controlBuffer, MSG_MAX_SIZE, mr_controlBuffer);

        //Post the Receive WQE
        post_RECEIVE_WQE(&controlRcvWqe);
        cout << "Waiting for first control message" << endl;
        //Wait For Completion - Print the Initial Message
        ibv_wc wc;
        PollCQ(&wc);


        Message* m = new Message(1, 0, MSG_MAX_SIZE, controlBuffer);
        cout << "Established Connection with Sensor printing control message expect all 0xFF" << endl;
        m->printBuffer(32);

    }



}

RdmaUdTransport::~RdmaUdTransport() {
    //Clean the RDMA Contexts
    CleanUpCMContext();
    CleanUpQPContext();

    //REmove the Shared MEmory
    delete mr_controlBuffer;
}

int RdmaUdTransport::push(Message* m)
{
    DEBUG("Sent a Msg: " << *m << endl);

    ibv_mr* mr_msg = create_MEMORY_REGION(&m->buffer, m->bufferSize);
    initSendWqe(&controlSendWqe, 42);
    updateSendWqe(&controlSendWqe, &(m->buffer), m->bufferSize, mr_msg);
    post_SEND_WQE(&controlSendWqe);

    DEBUG("DEBUG: Sent Message:\n");
    #ifdef DEBUG_BUILD
        m->printBuffer(32);
    #endif

    //Wait For Completion
    ibv_wc wc;
    int ret = 0;

    DEBUG("DEBUG: Waiting for CQE\n");
    do {
        ret = ibv_poll_cq(g_cq, 1, &wc);
    } while(ret == 0);
    DEBUG("DEBUG: Received " << ret << " CQE Elements\n");
    DEBUG("DEBUG: WRID(" << wc->wr_id << ")\tStatus(" << wc->status << ")\n");

    if(wc.status == IBV_WC_RNR_RETRY_EXC_ERR)
    {
        usleep(50); //wait 50 us and we will try again.
        cerr << "DEBUG: WRID(" << wc.wr_id << ")\tStatus(IBV_WC_RNR_RETRY_EXC_ERR)" << endl;
        return -1;
    }
    if(wc.status != IBV_WC_SUCCESS)
    {
        cerr << "DEBUG: WRID(" << wc.wr_id << ")\tStatus(" << wc.status << ")" << endl;
        return -1;
    }

   usleep(1000); //wait 50 us and we will try again.

    return 0;
}

/*
*  Pulls messages from the transport and places it in the buffer
*/
int RdmaUdTransport::pop(Message* m, int numReqMsg, int& numRetMsg, eTransportDest dest)
{
    numRetMsg = 0;

    do {
        //Post the RcvWQE
        post_RECEIVE_WQE(&controlRcvWqe);

        int r = 0;
        DEBUG("DEBUG: Waiting for CQE\n");
        do {
            r = ibv_poll_cq(g_cq, 1, &controlWc);
        } while (r == 0);
        DEBUG("DEBUG: Received " << r << " CQE Elements\n");

        numRetMsg += r;

        for (int j = 0; j < r; j++) {
            DEBUG ("test");
            DEBUG("DEBUG: WRID(" << controlWc.wr_id <<
                                 ")\tStatus(" << controlWc.status << ")" <<
                                 ")\tSize(" << controlWc.byte_len << ")\n");
        }

        m[numRetMsg-1] = Message(numRetMsg-1, 0, controlWc.byte_len, controlBuffer); //we can reuse the buffer now.
        //TODO: Choose to create message buffer in GPU vs CPU Memory.

        DEBUG ("DEBUG: Received Message:\n");
        #ifdef DEBUG_BUILD
        m[numRetMsg-1].printBuffer(32);
        #endif

    } while(numRetMsg < numReqMsg);

    return 0;
}

/*
 * Returns -1 for error otherwise return Number of Completions Received
 */
int RdmaUdTransport::PollCQ(ibv_wc* wc)
{
    int ret = 0;

    DEBUG("DEBUG: Waiting for CQE\n");
    do {
        ret = ibv_poll_cq(g_cq, 1, wc);
    } while(ret == 0);
    DEBUG("DEBUG: Received " << ret << " CQE Elements\n");
    DEBUG("DEBUG: WRID(" << wc->wr_id << ")\tStatus(" << wc->status << ")\n");
    return ret;
}

int RdmaUdTransport::initSendWqe(ibv_send_wr* wqe, int i)
{
    struct ibv_sge *sge;

    //wqe = (ibv_send_wr *)malloc(sizeof(ibv_send_wr));
    sge = (ibv_sge *)malloc(sizeof(ibv_sge));

    //memset(wqe, 0, sizeof(ibv_send_wr));
    memset(sge, 0, sizeof(ibv_sge));

    wqe->wr_id = i;
    wqe->opcode = IBV_WR_SEND;
    wqe->sg_list = sge;
    wqe->num_sge = 1;
    wqe->send_flags = IBV_SEND_SIGNALED;

    return 0;
}

int RdmaUdTransport::updateSendWqe(ibv_send_wr* wqe, void* buffer, size_t bufferlen, ibv_mr* bufferMemoryRegion)
{
    wqe->sg_list->addr = (uintptr_t)buffer;
    wqe->sg_list->length = bufferlen;
    wqe->sg_list->lkey = bufferMemoryRegion->lkey;

    return 0;
}

int RdmaUdTransport::initRecvWqe(ibv_recv_wr* wqe, int id)
{
    struct ibv_sge *sge;

    sge = (ibv_sge *)malloc(sizeof(ibv_sge));

    memset(sge, 0, sizeof(ibv_sge));

    wqe->wr_id = id;
    wqe->next = NULL;
    wqe->sg_list = sge;
    wqe->num_sge = 1;

    return 0;
}

int RdmaUdTransport::updateRecvWqe(ibv_recv_wr *wqe, void *buffer, size_t bufferlen, ibv_mr *bufferMemoryRegion) {

    wqe->sg_list->addr = (uintptr_t)buffer;
    wqe->sg_list->length = bufferlen;
    wqe->sg_list->lkey = bufferMemoryRegion->lkey;
    return 0;
}

int RdmaUdTransport::post_SEND_WQE(ibv_send_wr* ll_wqe)
{
    int err;
    int ret = 0;
    struct ibv_send_wr *bad_wqe = NULL;

    err = ibv_post_send(g_CMId->qp, ll_wqe, &bad_wqe);
    while(err != 0)
    {
        fprintf(stderr,"ERROR: post_SEND_WQE Error %u\n", err);
        if(err == ENOMEM && ret++ < 10) //Queue Full Wait for CQ Polling Thread to Clear
        {
            fprintf(stderr,"ERROR: Send Queue Full Retry %u of 10\n", ret);
            usleep(100); //Wait 100 Microseconds, max of 1 msec
        }
        else
        {
            fprintf(stderr, "ERROR: Unrecoverable Send Queue, aborting\n");
            return -1;
        }
    }

    return 0;
}

int RdmaUdTransport::post_RECEIVE_WQE(ibv_recv_wr* ll_wqe)
{
    DEBUG("DEBUG: Enter post_RECEIVE_WQE\n");
    int ret = 0;
    struct ibv_recv_wr *bad_wqe = NULL;

    ret = ibv_post_recv(g_CMId->qp, ll_wqe, &bad_wqe);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: post_RECEIVE_WQE - Couldn't Post Receive WQE\n");
        return -1;
    }

    DEBUG("DEBUG: Exit post_RECEIVE_WQE\n");
    return 0;
}

ibv_mr* RdmaUdTransport::create_MEMORY_REGION(void* buffer, size_t bufferlen)
{
    ibv_mr* tmpmr = (ibv_mr*)malloc(sizeof(ibv_mr));
    int mr_flags = IBV_ACCESS_LOCAL_WRITE | IBV_ACCESS_REMOTE_READ | IBV_ACCESS_REMOTE_WRITE;
    tmpmr = ibv_reg_mr(g_pd, buffer, bufferlen, mr_flags);
    if(!tmpmr)
    {
        fprintf(stderr, "ERROR: create_MEMORY_REGION: Couldn't Register memory region\n");
        return NULL;
    }

#ifdef DEBUG_BUILD
    fprintf(stderr, "DEBUG: Memory Region was registered with addr=%p, lkey=0x%x, rkey=0x%x, flags=0x%x\n",
            buffer, tmpmr->lkey, tmpmr->rkey, mr_flags);
#endif

    return tmpmr;
}

int RdmaUdTransport::GetCMEvent(rdma_cm_event_type* EventType)
{
    int ret;
    struct rdma_cm_event *CMEvent;

    ret = rdma_get_cm_event(g_CMEventChannel, & CMEvent);
    if(ret != 0)
    {
        fprintf(stderr,"ERROR: No CM Event Received in Time Out\n");
        return -1;
    }
    *EventType = CMEvent->event;
    PrintCMEvent(CMEvent);

    /*
     * Release the Event now that we are done with it
     */
    ret=rdma_ack_cm_event(CMEvent);
    if(ret != 0)
    {
        fprintf(stderr,"ERROR: CM couldn't release CM Event\n");
        return -1;
    }

    return 0;

}

int RdmaUdTransport::RDMACreateChannel()
{
    int ret = 0;
    g_CMEventChannel = NULL;

    // Open a Channel to the Communication Manager used to receive async events from the CM.
    g_CMEventChannel = rdma_create_event_channel();
    if(!g_CMEventChannel)
    {
        fprintf(stderr,"ERROR: Failed to Open CM Event Channel");
        CleanUpCMContext();
        return -1;
    }

    ret = rdma_create_id(g_CMEventChannel,&g_CMId, NULL, RDMA_PS_TCP);
    if(ret != 0)
    {
        fprintf(stderr,"ERROR: Failed to Create CM ID");
        CleanUpCMContext();
        return -1;
    }

    return 0;
}

int RdmaUdTransport::RDMAClientInit()
{
    int ret;
    rdma_cm_event_type et;

    if(get_addr(s_srcAddr.c_str(), (struct sockaddr*)&g_srcAddr) != 0)
    {
        fprintf(stderr,"ERROR: Failed to Resolve Local Address\n");
        CleanUpCMContext();
        return -1;
    }


    if(get_addr(s_dstAddr.c_str(),(struct sockaddr*)&g_dstAddr) != 0)
    {
        fprintf(stderr,"ERROR: Failed to Resolve Destination Address\n");
        CleanUpCMContext();
        return -1;
    }
    g_dstAddr.sin_port = n_dstPort;
    char str[INET_ADDRSTRLEN];
    inet_ntop(AF_INET, &(g_dstAddr.sin_addr), str, INET_ADDRSTRLEN);
    fprintf(stderr,"Processor address(%s) port(%u)\n", str, g_dstAddr.sin_port);

    /*
     * Resolve the IP Addresses to GIDs.
     */
    fprintf(stderr,"DEBUG: Resolving IP addresses to GIDS ...\n");
    ret = rdma_resolve_addr(g_CMId, (struct sockaddr*)&g_srcAddr, (struct sockaddr*)&g_dstAddr,2000);
    if(ret != 0)
    {
        fprintf(stderr,"ERROR: CM couldn't resolve IP addresses to GIDS\n");
        return -1;
    }

    fprintf(stderr,"DEBUG: Waiting for CM to resolve IP Addresses ...\n");
    do
    {
        ret = GetCMEvent(&et);
        if(ret != 0)
        {
            fprintf(stderr,"ERROR: Failed processing CM Events\n");
        }
    } while(et != RDMA_CM_EVENT_ADDR_RESOLVED);

    return 0;
}

int RdmaUdTransport::RDMAServerInit()
{
    int ret;

    if(get_addr(s_srcAddr.c_str(),(struct sockaddr*)&g_srcAddr) != 0)
    {
        fprintf(stderr, "ERROR: Failed to Resolve Local Address\n");
        CleanUpCMContext();
        return -1;
    }
    g_srcAddr.sin_port = n_srcPort;

    ret = rdma_bind_addr(g_CMId, (struct sockaddr*)&g_srcAddr);
    if(ret != 0 )
    {
        fprintf(stderr, "ERROR: RDMAServerInit - Couldn't bind to local address\n");
    }

    rdma_listen(g_CMId, 10);

    uint16_t port = 0;
    port = rdma_get_src_port(g_CMId);
    fprintf(stderr, "DEBUG: Listening on port %d.\n", port);
    return 0;
}

int RdmaUdTransport::RDMACreateQP()
{
    int ret;
    struct ibv_qp_init_attr qp_init_attr;

    //Create a Protection Domain
    g_pd = ibv_alloc_pd(g_CMId->verbs);
    if(!g_pd)
    {
        fprintf(stderr,"ERROR: - RDMACreateQP: Couldn't allocate protection domain\n");
        return -1;
    }

    /*Create a completion Queue */
    g_cq = ibv_create_cq(g_CMId->verbs, NUM_OPERATIONS, NULL, NULL, 0);
    if(!g_cq)
    {
        fprintf(stderr, "ERROR: RDMACreateQP - Couldn't create completion queue\n");
        return -1;
    }

    /* create the Queue Pair */
    memset(&qp_init_attr, 0, sizeof(qp_init_attr));

    qp_init_attr.qp_type = IBV_QPT_RC;
    qp_init_attr.sq_sig_all = 0;
    qp_init_attr.send_cq = g_cq;
    qp_init_attr.recv_cq = g_cq;
    qp_init_attr.cap.max_send_wr = NUM_OPERATIONS;
    qp_init_attr.cap.max_recv_wr = NUM_OPERATIONS;
    qp_init_attr.cap.max_send_sge = 1;
    qp_init_attr.cap.max_recv_sge = 1;


    ret = rdma_create_qp(g_CMId, g_pd, &qp_init_attr);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: RDMACreateQP: Couldn't Create Queue Pair Error(%d)\n", errno);
        return -1;
    }
    return 0;
}

int RdmaUdTransport::RDMAClientConnect()
{
    int ret;
    rdma_cm_event_type et;

    //rdma resolve route
    ret = rdma_resolve_route(g_CMId, 2000);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: RDMAClientConnect: Couldn't resolve the Route\n");
        return -1;
    }

    fprintf(stderr, "DEBUG: Waiting for Resolve Route CM Event ...\n");
    do
    {
        ret = GetCMEvent(&et);
        if(ret != 0)
        {
            fprintf(stderr, "ERROR: Processing CM Events\n");
        }
    } while(et != RDMA_CM_EVENT_ROUTE_RESOLVED);

    fprintf(stderr, "DEBUG: Waiting for Connection Established Event ...\n");

    struct rdma_conn_param ConnectionParams;

    memset(&ConnectionParams, 0, sizeof(ConnectionParams));
    ret = rdma_connect(g_CMId, &ConnectionParams);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: Client Couldn't Establish Connection\n");
        return -1;
    }

    PrintConnectionInfo(ConnectionParams);

    do
    {
        ret = GetCMEvent(&et);
        if(ret != 0)
        {
            fprintf(stderr, "ERROR: Processing CM Events\n");
        }
    } while(et != RDMA_CM_EVENT_ESTABLISHED);



    return 0;
}

int RdmaUdTransport::RDMAServerConnect()
{
    int ret;
    struct rdma_cm_event *CMEvent;
    rdma_cm_event_type et;

    /*
     * Wait for the Connect REquest to Come From the Client
     */
    do
    {
        ret = rdma_get_cm_event(g_CMEventChannel, & CMEvent);
        if(ret != 0)
        {
            fprintf(stderr, "ERROR: No Event Received Time Out\n");
            return -1;
        }

        PrintCMEvent(CMEvent);
    } while(CMEvent->event != RDMA_CM_EVENT_CONNECT_REQUEST);

    /*
     * Get the CM Id from the Event
     */

    g_CMId = CMEvent->id;
    /*
     * Now we can create the QP
     */
    ret = RDMACreateQP();
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: RDMAServerConnect - Couldn't Create QP\n");
        return -1;
    }

    struct rdma_conn_param ConnectionParams;
    memset(&ConnectionParams, 0, sizeof(ConnectionParams));
    ret = rdma_accept(g_CMId, &ConnectionParams);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: Client Couldn't Establish Connection\n");
        return -1;
    }

    PrintConnectionInfo(ConnectionParams);

    /*
     * Release the Event now that we are done with it
     */
    ret=rdma_ack_cm_event(CMEvent);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: couldn't release CM Event\n");
        return -1;
    }

    fprintf(stderr, "DEBUG: Waiting for Connection Established Event ...\n");
    do
    {
        ret = GetCMEvent(&et);
        if(ret != 0)
        {
            fprintf(stderr, "ERROR: Processing CM Events\n");
        }
    } while(et != RDMA_CM_EVENT_ESTABLISHED);

    return 0;
}

void RdmaUdTransport::CleanUpCMContext()
{
    if(g_CMEventChannel != NULL)
    {
        rdma_destroy_event_channel(g_CMEventChannel);
    }

    if(g_CMId != NULL)
    {
        if(rdma_destroy_id(g_CMId) != 0)
        {
            fprintf(stderr, "ERROR: CleanUpCMContext - Failed to destroy Connection Manager Id\n");
        }
    }
}

void RdmaUdTransport::CleanUpQPContext()
{
    if(g_pd != NULL)
    {
        if(ibv_dealloc_pd(g_pd) != 0)
        {
            fprintf(stderr, "ERROR: CleanUpQPContext - Failed to destroy Protection Domain\n");
        }
    }

    if(g_cq != NULL)
    {
        ibv_destroy_cq(g_cq);
        {
            fprintf(stderr, "ERROR: CleanUpQPContext - Failed to destroy Completion Queue\n");
        }
    }

    rdma_destroy_qp(g_CMId);

}


