#include "hip/hip_runtime.h"
//
// Created by alex on 8/7/20.
//

#include "rdma_ud_transport.cuh"

#include <cstdio>
#include <algorithm>
#include <arpa/inet.h>
#include <iostream>
#include <sys/types.h>
#include <sys/socket.h>
#include <netdb.h>


int get_addr(const char *dst, struct sockaddr *addr)
{
    struct addrinfo *res;
    int ret;
    ret = getaddrinfo(dst, NULL, NULL, &res);
    if (ret)
    {
        fprintf(stderr,"ERROR: getaddrinfo failed - invalid hostname or IP address\n");
        return -1;
    }
    memcpy(addr, res->ai_addr, res->ai_addrlen);
    freeaddrinfo(res);
    return ret;
}

void PrintCMEvent(struct rdma_cm_event *event)
{
    if(event->event == RDMA_CM_EVENT_ADDR_RESOLVED)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ADDR_RESOLVED)\n");
    else if(event->event == RDMA_CM_EVENT_ADDR_RESOLVED)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ADDR_RESOLVED)\n");
    else if(event->event == RDMA_CM_EVENT_ROUTE_RESOLVED)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ROUTE_RESOLVED)\n");
    else if(event->event == RDMA_CM_EVENT_ROUTE_ERROR)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ROUTE_ERROR)\n");
    else if(event->event == RDMA_CM_EVENT_CONNECT_REQUEST)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_CONNECT_REQUEST)\n");
    else if(event->event == RDMA_CM_EVENT_CONNECT_RESPONSE)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_CONNECT_RESPONSE)\n");
    else if(event->event == RDMA_CM_EVENT_CONNECT_ERROR)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_CONNECT_ERROR)\n");
    else if(event->event == RDMA_CM_EVENT_UNREACHABLE)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_UNREACHABLE)\n");
    else if(event->event == RDMA_CM_EVENT_REJECTED)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_REJECTED) Status(%u)", event->status );
    else if(event->event == RDMA_CM_EVENT_ESTABLISHED)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ESTABLISHED)\n");
    else if(event->event == RDMA_CM_EVENT_DISCONNECTED)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_DISCONNECTED)\n");
    else if(event->event == RDMA_CM_EVENT_DEVICE_REMOVAL)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_DEVICE_REMOVAL)\n");
    else if(event->event == RDMA_CM_EVENT_MULTICAST_JOIN)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_MULTICAST_JOIN)\n");
    else if(event->event == RDMA_CM_EVENT_MULTICAST_ERROR)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_MULTICAST_ERROR)\n");
    else if(event->event == RDMA_CM_EVENT_ADDR_CHANGE)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_ADDR_CHANGE)\n");
    else if(event->event == RDMA_CM_EVENT_TIMEWAIT_EXIT)
        fprintf(stderr,"DEBUG: Received CM Event(RDMA_CM_EVENT_TIMEWAIT_EXIT)\n");
    return;
}

void PrintConnectionInfo(rdma_conn_param cParam)
{
    fprintf(stderr, "DEBUG: QPN(%d)\n", cParam.qp_num);
}

RdmaUdTransport::RdmaUdTransport(string localAddr, string mcastAddr, eTransportRole role) {

    ibv_wc wc;
    s_localAddr = localAddr;
    s_mcastAddr = mcastAddr;

    // Creating socket file descriptor
    if(RDMACreateContext() != 0)
    {
        cerr << "Failed Create the RDMA Channel." << endl;
        exit(EXIT_FAILURE);
    }

    if(role == eTransportRole::SENSOR) { //Sensor

        if(RDMACreateQP() != 0)
        {
            //TODO: We should be able to start the processes in any order. Need things to wait and retry.
            fprintf(stdout, "Exiting - Failed to Create Queue Pair, make sure processor is running\n");
            exit(EXIT_FAILURE);
        }

    } else { //Processor

        if(RDMACreateQP() != 0)
        {
            fprintf(stderr, "Exiting - Failed to establish connection with the client\n");
            exit(EXIT_FAILURE);
        }

    }

    if(RdmaMcastConnect() != 0)
    {
        fprintf(stdout, "Exiting - Failed to establish connection to MultiCast Group\n");
        exit(EXIT_FAILURE);
    }

    //Initialize the Data Channel
    mr_dataBuffer = create_MEMORY_REGION(&dataBuffer, MSG_MAX_SIZE);
    memset(dataBuffer, 0x00, MSG_MAX_SIZE);
    initSendWqe(&dataSendWqe, 0);
    updateSendWqe(&dataSendWqe, &dataBuffer, MSG_MAX_SIZE, mr_dataBuffer);
    initRecvWqe(&dataRcvWqe, 0);
    updateRecvWqe(&dataRcvWqe, &dataBuffer, MSG_MAX_SIZE, mr_dataBuffer);

    //Register the control plane memory region
    mr_controlBuffer = create_MEMORY_REGION(&controlBuffer, MSG_MAX_SIZE);
    memset(controlBuffer, 0x00, MSG_MAX_SIZE);
    initSendWqe(&controlSendWqe, 0);
    updateSendWqe(&controlSendWqe, &controlBuffer, MSG_MAX_SIZE, mr_controlBuffer);
    initRecvWqe(&controlRcvWqe, 0);
    updateRecvWqe(&controlRcvWqe, &controlBuffer, MSG_MAX_SIZE, mr_controlBuffer);

}

RdmaUdTransport::~RdmaUdTransport() {
    //Clean the RDMA Contexts
    DestroyContext();
    DestroyQP();

    //REmove the Shared MEmory
    delete mr_controlBuffer;
}

int RdmaUdTransport::push(Message* m)
{
    //usleep(200); //TODO: Take this out
    //cerr << "NO PUSH OP" << endl;

    ibv_mr* mr_msg = create_MEMORY_REGION(&m->buffer, m->bufferSize);

    initSendWqe(&dataSendWqe, 42);
    updateSendWqe(&dataSendWqe, &(m->buffer), m->bufferSize, mr_msg);

      post_SEND_WQE(&dataSendWqe);

       DEBUG("DEBUG: Sent Message:\n");
      #ifdef DEBUG_BUILD
          m->printBuffer(32);
      #endif

      //Wait For Completion
      int ret = 0;

      DEBUG("DEBUG: Waiting for CQE\n");
      do {
          ret = ibv_poll_cq(g_cq, 1, &dataWc);
      } while(ret == 0);
      DEBUG("DEBUG: Received " << ret << " CQE Elements\n");
      DEBUG("DEBUG: WRID(" << dataWc.wr_id << ")\tStatus(" << dataWc.status << ")\n");

      if(dataWc.status == IBV_WC_RNR_RETRY_EXC_ERR)
      {
          usleep(50); //wait 50 us and we will try again.
          cerr << "DEBUG: WRID(" << dataWc.wr_id << ")\tStatus(IBV_WC_RNR_RETRY_EXC_ERR)" << endl;
          ibv_dereg_mr(mr_msg);
          return -1;
      }
      if(dataWc.status != IBV_WC_SUCCESS)
      {
          cerr << "DEBUG: WRID(" << dataWc.wr_id << ")\tStatus(" << dataWc.status << ")" << endl;
          ibv_dereg_mr(mr_msg);
          return -1;
      }

    ibv_dereg_mr(mr_msg);



    return 0;
}

/*
*  Pulls messages from the transport and places it in the buffer
*/
int RdmaUdTransport::pop(Message* m, int numReqMsg, int& numRetMsg, eTransportDest dest)
{
    numRetMsg = 0;

    do {

        //Post the RcvWQE
        post_RECEIVE_WQE(&dataRcvWqe);

        int r = 0;
        DEBUG("DEBUG: Waiting for CQE\n");
        do {
            r = ibv_poll_cq(g_cq, 1, &dataWc);
        } while (r == 0);
        DEBUG("DEBUG: Received " << r << " CQE Elements\n");

        numRetMsg += r;

        for (int j = 0; j < r; j++) {
            DEBUG ("test");
            DEBUG("DEBUG: WRID(" << dataWc.wr_id <<
                                 ")\tStatus(" << dataWc.status << ")" <<
                                 ")\tSize(" << dataWc.byte_len << ")\n");
        }

        m[numRetMsg-1] = Message(numRetMsg-1, 0, dataWc.byte_len, dataBuffer); //we can reuse the buffer now.
        //TODO: Choose to create message buffer in GPU vs CPU Memory.

        DEBUG ("DEBUG: Received Message:\n");
        #ifdef DEBUG_BUILD
        m[numRetMsg-1].printBuffer(32);
        #endif

    } while(numRetMsg < numReqMsg);

    return 0;
}

/*
 * Returns -1 for error otherwise return Number of Completions Received
 */
int RdmaUdTransport::PollCQ(ibv_wc* wc)
{
    int ret = 0;

    DEBUG("DEBUG: Waiting for CQE\n");
    do {
        ret = ibv_poll_cq(g_cq, 1, wc);
    } while(ret == 0);
    DEBUG("DEBUG: Received " << ret << " CQE Elements\n");
    DEBUG("DEBUG: WRID(" << wc->wr_id << ")\tStatus(" << wc->status << ")\n");
    return ret;
}

int RdmaUdTransport::initSendWqe(ibv_send_wr* wqe, int i)
{
    struct ibv_sge *sge;

    //wqe = (ibv_send_wr *)malloc(sizeof(ibv_send_wr));
    sge = (ibv_sge *)malloc(sizeof(ibv_sge));

    //memset(wqe, 0, sizeof(ibv_send_wr));
    memset(sge, 0, sizeof(ibv_sge));

    wqe->wr_id = i;
    wqe->opcode = IBV_WR_SEND;
    wqe->sg_list = sge;
    wqe->num_sge = 1;
    wqe->send_flags = IBV_SEND_SIGNALED;

    wqe->wr.ud.ah = AddressHandle;
    wqe->wr.ud.remote_qpn = RemoteQpn;
    wqe->wr.ud.remote_qkey = RemoteQkey;

    return 0;
}

int RdmaUdTransport::updateSendWqe(ibv_send_wr* wqe, void* buffer, size_t bufferlen, ibv_mr* bufferMemoryRegion)
{
    wqe->sg_list->addr = (uintptr_t)buffer;
    wqe->sg_list->length = bufferlen;
    wqe->sg_list->lkey = bufferMemoryRegion->lkey;
    return 0;
}

int RdmaUdTransport::initRecvWqe(ibv_recv_wr* wqe, int id)
{
    struct ibv_sge *sge;

    sge = (ibv_sge *)malloc(sizeof(ibv_sge));

    memset(sge, 0, sizeof(ibv_sge));

    wqe->wr_id = id;
    wqe->next = NULL;
    wqe->sg_list = sge;
    wqe->num_sge = 1;

    return 0;
}

int RdmaUdTransport::updateRecvWqe(ibv_recv_wr *wqe, void *buffer, size_t bufferlen, ibv_mr *bufferMemoryRegion) {

    wqe->sg_list->addr = (uintptr_t)buffer;
    wqe->sg_list->length = bufferlen;
    wqe->sg_list->lkey = bufferMemoryRegion->lkey;
    return 0;
}

int RdmaUdTransport::post_SEND_WQE(ibv_send_wr* ll_wqe)
{
    int err;
    int ret = 0;
    struct ibv_send_wr *bad_wqe = NULL;

    err = ibv_post_send(g_CMId->qp, ll_wqe, &bad_wqe);
    while(err != 0)
    {
        fprintf(stderr,"ERROR: post_SEND_WQE Error %u\n", err);
        if(err == ENOMEM && ret++ < 10) //Queue Full Wait for CQ Polling Thread to Clear
        {
            fprintf(stderr,"ERROR: Send Queue Full Retry %u of 10\n", ret);
            usleep(100); //Wait 100 Microseconds, max of 1 msec
        }
        else
        {
            fprintf(stderr, "ERROR: Unrecoverable Send Queue, aborting\n");
            return -1;
        }
    }

    return 0;
}

int RdmaUdTransport::post_RECEIVE_WQE(ibv_recv_wr* ll_wqe)
{
    DEBUG("DEBUG: Enter post_RECEIVE_WQE\n");
    int ret = 0;
    struct ibv_recv_wr *bad_wqe = NULL;

    ret = ibv_post_recv(g_CMId->qp, ll_wqe, &bad_wqe);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: post_RECEIVE_WQE - Couldn't Post Receive WQE\n");
        return -1;
    }

    DEBUG("DEBUG: Exit post_RECEIVE_WQE\n");
    return 0;
}

ibv_mr* RdmaUdTransport::create_MEMORY_REGION(void* buffer, size_t bufferlen)
{
    ibv_mr* tmpmr = (ibv_mr*)malloc(sizeof(ibv_mr));
    //int mr_flags = IBV_ACCESS_LOCAL_WRITE | IBV_ACCESS_REMOTE_READ | IBV_ACCESS_REMOTE_WRITE;
    int mr_flags = IBV_ACCESS_LOCAL_WRITE;
    tmpmr = ibv_reg_mr(g_pd, buffer, bufferlen, mr_flags);
    if(!tmpmr)
    {
        fprintf(stderr, "ERROR: create_MEMORY_REGION: Couldn't Register memory region\n");
        return NULL;
    }

#ifdef DEBUG_BUILD
    fprintf(stderr, "DEBUG: Memory Region was registered with addr=%p, lkey=0x%x, rkey=0x%x, flags=0x%x\n",
            buffer, tmpmr->lkey, tmpmr->rkey, mr_flags);
#endif

    return tmpmr;
}

int RdmaUdTransport::GetCMEvent(rdma_cm_event_type* EventType)
{
    int ret;
    struct rdma_cm_event *CMEvent;

    ret = rdma_get_cm_event(g_CMEventChannel, & CMEvent);
    if(ret != 0)
    {
        fprintf(stderr,"ERROR: No CM Event Received in Time Out\n");
        return -1;
    }
    *EventType = CMEvent->event;
    PrintCMEvent(CMEvent);

    /*
     * Release the Event now that we are done with it
     */
    ret=rdma_ack_cm_event(CMEvent);
    if(ret != 0)
    {
        fprintf(stderr,"ERROR: CM couldn't release CM Event\n");
        return -1;
    }

    return 0;

}

/*
 * Create the CM Event Channel, the Connection Identifier, Bind the application to a local address
 */
int RdmaUdTransport::RDMACreateContext()
{
    int ret = 0;
    struct rdma_cm_event *CMEvent;

    // Open a Channel to the Communication Manager used to receive async events from the CM.
    g_CMEventChannel = rdma_create_event_channel();
    if(!g_CMEventChannel)
    {
        fprintf(stderr,"ERROR - RDMACreateContext: Failed to Create CM Event Channel");
        DestroyContext();
        return -1;
    }

    ret = rdma_create_id(g_CMEventChannel, &g_CMId, NULL, RDMA_PS_UDP);
    if(ret != 0)
    {
        fprintf(stderr,"ERROR - RDMACreateContext: Failed to Create CM ID");
        DestroyContext();
        return -1;
    }

    if(get_addr(s_localAddr.c_str(), (struct sockaddr*)&localAddr_in) != 0)
    {
        fprintf(stderr, "ERROR - RDMACreateContext: Failed to Resolve Local Address\n");
        DestroyContext();
        return -1;
    }

    if(get_addr(s_mcastAddr.c_str(), (struct sockaddr*)&mcastAddr_in) != 0)
    {
        fprintf(stderr, "ERROR - RDMACreateContext: Failed to Resolve Multicast Address Address\n");
        DestroyContext();
        return -1;
    }

    ret = rdma_bind_addr(g_CMId, (struct sockaddr*)&localAddr_in);
    if(ret != 0 )
    {
        fprintf(stderr, "ERROR - RDMACreateContext: Couldn't bind to local address\n");
        fprintf(stderr, "ERROR - errno %s\n", strerror(errno));
        return -1;
    }

    ret = rdma_resolve_addr(g_CMId,
                            (struct sockaddr*)&localAddr_in,
                            (struct sockaddr*)&mcastAddr_in,
                            2000);
    if(ret != 0 )
    {
        fprintf(stderr, "ERROR - RDMACreateContext: Couldn't resolve local address and or mcast address.\n");
        fprintf(stderr, "ERROR - errno %s\n", strerror(errno));
        return -1;
    }

    ret = rdma_get_cm_event(g_CMEventChannel, &CMEvent);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR - RDMACreateContext: No Event Received Time Out\n");
        return -1;
    }
    if(CMEvent->event != RDMA_CM_EVENT_ADDR_RESOLVED)
    {
        fprintf(stderr, "ERROR - RDMACreateContext: Expected Multicast Joint Event\n");
        return -1;
    }


    return 0;
}

int RdmaUdTransport::RDMACreateQP()
{
    int ret;
    struct ibv_qp_init_attr qp_init_attr;

    //g_CMId->qp_type = IBV_QPT_UD;
    //g_CMId->ps = RDMA_PS_UDP;

    //Create a Protection Domain
    g_pd = ibv_alloc_pd(g_CMId->verbs);
    if(!g_pd)
    {
        fprintf(stderr,"ERROR: - RDMACreateQP: Couldn't allocate protection domain\n");
        fprintf(stderr, "ERROR - errno %s\n", strerror(errno));
        return -1;
    }

    /*Create a completion Queue */
    //g_cq = ibv_create_cq(g_CMId->verbs, NUM_OPERATIONS, NULL, NULL, 0);
    g_cq = ibv_create_cq(g_CMId->verbs, 5, NULL, NULL, 1);
    if(!g_cq)
    {
        fprintf(stderr, "ERROR: RDMACreateQP - Couldn't create completion queue\n");
        fprintf(stderr, "ERROR - errno %s\n", strerror(errno));
        return -1;
    }

    /* create the Queue Pair */
    memset(&qp_init_attr, 0, sizeof(qp_init_attr));

    qp_init_attr.qp_type = IBV_QPT_UD;
    //qp_init_attr.sq_sig_all = 0;
    qp_init_attr.send_cq = g_cq;
    qp_init_attr.recv_cq = g_cq;
    qp_init_attr.cap.max_send_wr = NUM_OPERATIONS;
    qp_init_attr.cap.max_recv_wr = NUM_OPERATIONS;
    qp_init_attr.cap.max_send_sge = 1;
    qp_init_attr.cap.max_recv_sge = 1;

    ret = rdma_create_qp(g_CMId, g_pd, &qp_init_attr);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: RDMACreateQP: Couldn't Create Queue Pair Error\n");
        fprintf(stderr, "ERROR - errno %s\n", strerror(errno));
        return -1;
    }
    return 0;
}

int RdmaUdTransport::RdmaMcastConnect()
{
    int ret = 0;
    struct rdma_cm_event *CMEvent;

    ret = rdma_join_multicast(g_CMId, (struct sockaddr*)&mcastAddr_in, NULL);
    if(ret)
    {
        fprintf(stderr, "RDMA multicast join Failed\n");
        fprintf(stderr, "ERROR - errno %s\n", strerror(errno));
        return -1;
    }

    ret = rdma_get_cm_event(g_CMEventChannel, &CMEvent);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: No Event Received Time Out\n");
        fprintf(stderr, "ERROR - errno %s\n", strerror(errno));
        return -1;
    }
    if(CMEvent->event == RDMA_CM_EVENT_MULTICAST_JOIN)
    {
        rdma_ud_param *param;
        param = &CMEvent->param.ud;

        RemoteQpn = param->qp_num;
        RemoteQkey = param->qkey;
        AddressHandle = ibv_create_ah(g_pd, &param->ah_attr);
        if (!AddressHandle)
        {
            fprintf(stderr, "ERROR OnMulticastJoin - Failed to create the Address Handle\n");
            return -1;
        }
        fprintf(stderr, "Joined Multicast Group QPN(%d) QKey(%d)\n", RemoteQpn, RemoteQkey);
    } else {

        fprintf(stderr, "Expected Multicast Joint Event\n");
        return -1;
    }



    return 0;
}

int RdmaUdTransport::RDMAClientConnect()
{
    int ret;
    rdma_cm_event_type et;

    //rdma resolve route
    ret = rdma_resolve_route(g_CMId, 2000);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: RDMAClientConnect: Couldn't resolve the Route\n");
        return -1;
    }

    fprintf(stderr, "DEBUG: Waiting for Resolve Route CM Event ...\n");
    do
    {
        ret = GetCMEvent(&et);
        if(ret != 0)
        {
            fprintf(stderr, "ERROR: Processing CM Events\n");
        }
    } while(et != RDMA_CM_EVENT_ROUTE_RESOLVED);

    fprintf(stderr, "DEBUG: Waiting for Connection Established Event ...\n");

    struct rdma_conn_param ConnectionParams;

    memset(&ConnectionParams, 0, sizeof(ConnectionParams));
    ret = rdma_connect(g_CMId, &ConnectionParams);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: Client Couldn't Establish Connection\n");
        return -1;
    }

    PrintConnectionInfo(ConnectionParams);

    do
    {
        ret = GetCMEvent(&et);
        if(ret != 0)
        {
            fprintf(stderr, "ERROR: Processing CM Events\n");
        }
    } while(et != RDMA_CM_EVENT_ESTABLISHED);



    return 0;
}

int RdmaUdTransport::RDMAServerConnect()
{
    int ret;
    struct rdma_cm_event *CMEvent;
    rdma_cm_event_type et;

    /*
     * Wait for the Connect REquest to Come From the Client
     */
    do
    {
        ret = rdma_get_cm_event(g_CMEventChannel, & CMEvent);
        if(ret != 0)
        {
            fprintf(stderr, "ERROR: No Event Received Time Out\n");
            return -1;
        }

        PrintCMEvent(CMEvent);
    } while(CMEvent->event != RDMA_CM_EVENT_CONNECT_REQUEST);

    /*
     * Get the CM Id from the Event
     */

    g_CMId = CMEvent->id;
    /*
     * Now we can create the QP
     */
    ret = RDMACreateQP();
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: RDMAServerConnect - Couldn't Create QP\n");
        return -1;
    }

    struct rdma_conn_param ConnectionParams;
    memset(&ConnectionParams, 0, sizeof(ConnectionParams));
    ret = rdma_accept(g_CMId, &ConnectionParams);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: Client Couldn't Establish Connection\n");
        return -1;
    }

    PrintConnectionInfo(ConnectionParams);

    /*
     * Release the Event now that we are done with it
     */
    ret=rdma_ack_cm_event(CMEvent);
    if(ret != 0)
    {
        fprintf(stderr, "ERROR: couldn't release CM Event\n");
        return -1;
    }

    fprintf(stderr, "DEBUG: Waiting for Connection Established Event ...\n");
    do
    {
        ret = GetCMEvent(&et);
        if(ret != 0)
        {
            fprintf(stderr, "ERROR: Processing CM Events\n");
        }
    } while(et != RDMA_CM_EVENT_ESTABLISHED);

    return 0;
}

void RdmaUdTransport::DestroyContext()
{
    if(g_CMEventChannel != NULL)
    {
        rdma_destroy_event_channel(g_CMEventChannel);
    }

    if(g_CMId != NULL)
    {
        if(rdma_destroy_id(g_CMId) != 0)
        {
            fprintf(stderr, "ERROR: DestroyContext - Failed to destroy Connection Manager Id\n");
        }
    }
}

void RdmaUdTransport::DestroyQP()
{
    if(g_pd != NULL)
    {
        if(ibv_dealloc_pd(g_pd) != 0)
        {
            fprintf(stderr, "ERROR: DestroyQP - Failed to destroy Protection Domain\n");
        }
    }

    if(g_cq != NULL)
    {
        ibv_destroy_cq(g_cq);
        {
            fprintf(stderr, "ERROR: DestroyQP - Failed to destroy Completion Queue\n");
        }
    }

    rdma_destroy_qp(g_CMId);

}


