#include "hip/hip_runtime.h"
//
// Created by alex on 7/15/20.
//

#include "Processor.cuh"

#define MSG_BLOCK_SIZE 1000

/*
__global__ void count_zeros(Message* msg, int* sum)
{
    int i = threadIdx.x;

    for(int j = 0; j < msg->bufferSize; i++)
    {
        if(msg->buffer[j] == 0)
            sum++;
    }
}
 */

void cpu_count_zeros(vector<Message*>& flow, int& sum)
{
    for(int i = 0; i < flow.size(); i++)
    {
        for(int j = 0; j < flow[i]->bufferSize; j++)
        {
            if(flow[i]->buffer[j] == 0)
            {
                sum += 1;
                //cout << "found a zero at msg[" << i << "] byte[" << j << "]" << endl;
            }
        }
    }
}


Processor::Processor(Transport* t) {
    transport = t;
}

int Processor::procCountZerosCPU(int minMessageToProcess) {
    chrono::time_point<chrono::system_clock> start;
    chrono::duration<double> timeToProcess;

    vector<Message> m;
    int r = 0;
    int sum = 0;
    int processedMessages = 0;

    start = chrono::system_clock::now();
    while (processedMessages < minMessageToProcess) {

        if (0 != transport->pop(m, MSG_BLOCK_SIZE, r)) {
            exit(EXIT_FAILURE);
        }

        if(r > 0) //If there are new messages process them
        {
            //cpu_count_zeros(m, sum);
            processedMessages += r;
        }
        //m.clear();
        r=0;

    }
    timeToProcess = chrono::system_clock::now() - start;

    cout << "Processing Completed: " << endl;
    cout << "\t processed " << processedMessages << " in " << timeToProcess.count() << " sec" << endl;
    cout << "\t total zero's in messages = " << sum << endl;
    exit(EXIT_SUCCESS);
}

int Processor::procPrintMessages(int minMessageToProcess) {
    vector<Message> m;
    int r = 0;

    while (r < minMessageToProcess) {
        if (0 != transport->pop(m, MSG_BLOCK_SIZE, r)) {
            exit(EXIT_FAILURE);
        }
    }

    //Simple process (i.e. print)
    cout << "Processing Completed: found " << r << "messages" << endl;
    for(int i = 0; i<r; i++)
    {
        m[i].printBuffer(32);
    }

    exit(EXIT_SUCCESS);
}
