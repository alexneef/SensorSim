#include "hip/hip_runtime.h"
//
// Created by alex on 7/15/20.
//

#include "Processor.cuh"

inline hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

__global__ void gpu_count_zeros(Message* flow, int* sum, int flowLength)
{
    int indx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = indx; i < flowLength; i += stride)
    {
        for(int j = 0; j < flow[i].bufferSize; j++)
        {
            if(flow[i].buffer[j] == 0)
            {
                sum[i] += 1;
                //cout << "found a zero at msg[" << i << "] byte[" << j << "]" << endl;
            }
        }
    }
}


void cpu_count_zeros(Message* flow, int& sum, int flowLength)
{
    for(int i = 0; i < flowLength; i++)
    {
        for(int j = 0; j < flow[i].bufferSize; j++)
        {
            if(flow[i].buffer[j] == 0)
            {
                sum += 1;
                //cout << "found a zero at msg[" << i << "] byte[" << j << "]" << endl;
            }
        }
    }
}


Processor::Processor(ITransport* t) {
    transport = t;
}

void Processor::procCountZerosGPU(int minMessageToProcess) {
    chrono::time_point<chrono::system_clock> start;
    chrono::duration<double> timeToProcess;

    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    size_t threadsPerBlock;
    size_t numberOfBlocks;

    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;

    int msgCountReturned = 0;
    int processedMessages = 0;
    int sum =0;

    Message* m;//Create array that is max message block size
    size_t msgBlockSize = MSG_BLOCK_SIZE * sizeof(Message);
    checkCuda( hipMallocManaged(&m, msgBlockSize));

    int* blockSum;   //Array with sum of zeros for this message
    size_t sumArraySize = MSG_BLOCK_SIZE * sizeof(int);
    checkCuda( hipMallocManaged(&blockSum, sumArraySize));
   // cout << "Processing on GPU using " <<  numberOfBlocks << " blocks with " << threadsPerBlock << " threads per block" << endl;

    start = chrono::system_clock::now();
    while (processedMessages < minMessageToProcess) {

        if (0 != transport->pop(m, MSG_BLOCK_SIZE, msgCountReturned, eTransportDest::DEVICE)) {
            exit(EXIT_FAILURE);
        }

        hipMemPrefetchAsync(m, msgBlockSize, deviceId);

        if(msgCountReturned > 0) //If there are new messages process them
        {
            cerr << "\rProcessed " << processedMessages << " messages";
            gpu_count_zeros <<< threadsPerBlock, numberOfBlocks >>>(m, blockSum, msgCountReturned);

            checkCuda( hipGetLastError() );
            checkCuda( hipDeviceSynchronize() ); //Wait for GPU threads to complete

            hipMemPrefetchAsync(blockSum, sumArraySize, hipCpuDeviceId);

            for(int k = 0; k < msgCountReturned; k++)
            {
                sum += blockSum[k]; //Add all the counts to the accumulator
                blockSum[k] = 0;
            }

            processedMessages += msgCountReturned;
        }
        //m.clear();
        msgCountReturned=0;

    }
    timeToProcess = chrono::system_clock::now() - start;

    checkCuda( hipFree(m));
    checkCuda( hipFree(blockSum));

    cout << "\n Processing Completed: " << endl;
    cout << "\t processed " << processedMessages << " in " << timeToProcess.count() << " sec" << endl;
    cout << "\t total zero's in messages = " << sum << endl;
    exit(EXIT_SUCCESS);
}

int Processor::procCountZerosCPU(int minMessageToProcess) {
    chrono::time_point<chrono::system_clock> start;
    chrono::duration<double> timeToProcess;

    Message m[MSG_BLOCK_SIZE];
    int msgCountReturned = 0;
    int sum = 0;
    int processedMessages = 0;

    start = chrono::system_clock::now();
    while (processedMessages < minMessageToProcess) {

        if (0 != transport->pop(m, MSG_BLOCK_SIZE, msgCountReturned, eTransportDest::HOST)) {
            exit(EXIT_FAILURE);
        }

        if(msgCountReturned > 0) //If there are new messages process them
        {
            cerr << "\rProcessed " << processedMessages << " messages";
            cpu_count_zeros(m, sum, msgCountReturned);
            processedMessages += msgCountReturned;
        }
        msgCountReturned=0;

    }
    timeToProcess = chrono::system_clock::now() - start;

    cout << "\nProcessing Completed: " << endl;
    cout << "\t processed " << processedMessages << " in " << timeToProcess.count() << " sec" << endl;
    cout << "\t total zero's in messages = " << sum << endl;
    exit(EXIT_SUCCESS);
}

int Processor::procPrintMessages(int minMessageToProcess) {
    Message m[MSG_BLOCK_SIZE];
    int processedCount = 0;
    int r = 0;

    do {

        if (0 != transport->pop(m, MSG_BLOCK_SIZE, r, eTransportDest::HOST)) {
            exit(EXIT_FAILURE);
        }

        processedCount += r;

        cout << "Printing first bytes of " << min(r,minMessageToProcess) << " messages" << endl;
        for(int i = 0; i<min(r,minMessageToProcess); i++)
        {
            m[i].printBuffer(32);
            cout << endl;
        }
    } while (processedCount < minMessageToProcess);

    //Simple process (i.e. print)
    cout << "Processing Completed: found " << processedCount << " messages" << endl;




    exit(EXIT_SUCCESS);
}
